#include "MyLittleCUDAWrapper.cuh"
#include "ErrorException.hpp"



void
cudaWrp::callAPI(hipError_t status, std::string_view name) {
	if(status != hipSuccess) {
		throw cudaWrp::ErrorException(status, name);
	}
}

void
cudaWrp::callAPI(hipError_t status, std::string_view name, const Params &p) {
	if(status != hipSuccess) {
		throw cudaWrp::ErrorException(status, name, p);
	}
}

hipError_t
cudaWrp::callKernel(std::function<void(void)>kernel, std::string_view name, const Params &p, bool synchronize) {
	hipError_t status = hipError_t::hipErrorUnknown;

	kernel();

	if(synchronize) {
		// Check for any errors launching the kernel
		callAPI(status = hipGetLastError(), "hipGetLastError", p);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		callAPI(status = hipDeviceSynchronize(), "hipDeviceSynchronize", p);
	}

	return status;
}

//must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
void cudaWrp::destroyContext() {
	callAPI(hipDeviceReset(), "hipDeviceReset");
}
